#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include "../deps/CudaCommon/hip/hip_vector_types.h"

hipError_t P2G_with_Cuda(// PARTICLES
    double2* particles_x, double pm,

    // GRID
    double* grid_m, double grid_dx, double2 grid_min_point, int grid_max_i, int grid_max_j);


__device__ double CubicBSpline(double x)
{
    x = abs(x);
    if (0.0 <= x && x < 1.0) {
        return 0.5 * x * x * x - x * x + 2.0 / 3.0;
    }
    else if (1.0 <= x && x < 2.0) {
        return (2.0 - x) * (2.0 - x) * (2.0 - x) / 6.0;
    }
    else {
        return 0.0;
    }
}

__global__ void P2G_Kernel(
    // PARTICLES
    double2* particles_x, double pm, 
    
    // GRID
    double* grid_m, double grid_dx, double2 grid_min_point, int grid_max_i, int grid_max_j)
{
    // Assume grid_m has been set to 0 for all indices


    int p_id = threadIdx.x;

    double2 px = particles_x[p_id];

    double2 relative_point;
    relative_point.x = px.x - grid_min_point.x;
    relative_point.y = px.y - grid_min_point.y;

    int bot_left_index_i = (int)floor(relative_point.x / grid_dx) - 1;
    int bot_left_index_j = (int)floor(relative_point.y / grid_dx) - 1;

    // CUBIC B-SPLINE
    for (int i = 0; i <= 3; i++) 
    {
        for (int j = 0; j <= 3; j++) 
        {
            // conditional branching... thats ok right?
            bool in_bounds = 0 < bot_left_index_i && bot_left_index_i < grid_max_i &&
                0 < bot_left_index_j && bot_left_index_j < grid_max_j;
            

            if (in_bounds)
            {
                grid_m[bot_left_index_i * grid_max_j + bot_left_index_j] += pm;
            }
        }
    }
}

__global__ void Grid_Reset_Kernel(double* grid_m, int grid_max_i, int grid_max_j)
{
    int i = threadIdx.x;
    int j = threadIdx.y;
    grid_m[i * grid_max_j + j] = 0;
}



int main()
{
    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to project particle masses onto the grid
hipError_t P2G_with_Cuda(
    // PARTICLES
    double2* particles_x, double pm,

    // GRID
    double* grid_m, double grid_dx, double2 grid_min_point, int grid_max_i, int grid_max_j)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
